﻿#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#include <stdio.h>
#define N 256
#define num_blocks 2
#define num_threads 128

__global__ void matrix_vector_multi_gpu(float* A_d, float* B_d, float* C_d) {
	int i, j;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	A_d[j] = 0.0F;
	for (i = 0; i < N; i++) {
		A_d[j] += B_d[j * N + i] * C_d[i];
	}
}

int main()
{
    int i, j;
	float A[N], B[N*N], C[N]; // Device
	float* A_d, * B_d, * C_d; // Host copies of A, B, C

	dim3 blocks(num_blocks, 1, 1);
	dim3 theads(num_threads, 1, 1);

    for (j = 0; j < N; j++) {
		for (i = 0; i < N; i++) {
			B[j * N + i] = ((float)j) / N;
		}
        C[j] = 1.0F;
    }

	// Memory allocation for device copies of A, B, C
	hipMalloc((void**)&A_d, N * sizeof(float));
	hipMalloc((void**)&B_d, N * N * sizeof(float));
	hipMalloc((void**)&C_d, N * sizeof(float));

	// Copy inputs to device
	hipMemcpy(A_d, A, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(C_d, C, N * sizeof(float), hipMemcpyHostToDevice);

	matrix_vector_multi_gpu <<< blocks, theads >>> (A_d, B_d, C_d);
	hipMemcpy(A, A_d, N * sizeof(float), hipMemcpyDeviceToHost);

    for (j = 0; j < N; j++) {
		printf("%f\n", A[j]);
    }

	// Free device memory
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return 0;
}
